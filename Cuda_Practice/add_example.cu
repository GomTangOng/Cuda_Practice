#include "hip/hip_runtime.h"
#include "stdafx.h"

__global__ void Add(int *a, int *b, int *c)
{
	int tid = threadIdx.x;

	if (tid < N)
		c[tid] = a[tid] + b[tid];
}

extern "C" void Cuda_Sum_Init(int *a, int *b, int *c)
{
	int *d_a, *d_b, *d_c;

	hipMalloc((void **)&d_a, sizeof(int) * N);
	hipMalloc((void **)&d_b, sizeof(int) * N);
	hipMalloc((void **)&d_c, sizeof(int) * N);

	for (int i = 0; i < N; ++i)
	{
		a[i] = i;
		b[i] = i * i;
		c[i] = 0;
	}

	hipMemcpy(d_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, sizeof(int) * N, hipMemcpyHostToDevice);

	Add << <1, N >> >(d_a, d_b, d_c);

	hipMemcpy(c, d_c, sizeof(int) * N, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}
