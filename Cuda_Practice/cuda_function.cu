#include "hip/hip_runtime.h"
#include "stdafx.h"

struct Sphere
{
	int radius;
	int x, y, z;

	__device__ float hit(float ox, float oy, float *n)
	{
		float dx = ox - x;
		float dy = oy - y;
		if (dx * dx + dy * dy < radius * radius)
		{
			float dz = sqrtf(radius * radius - dx * dx - dy * dy);
			*n = dz / sqrtf(radius * radius);
			return dz + z;
		}
		return -INFINITY;
	}
};



__global__ void Zero_Dot(float *a, float *b, float *c)
{
	__shared__ float cache[ThreadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;
	float temp = 0;
	while (tid < N) {
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}
	cache[cacheIndex] = temp;
	__syncthreads();

	int i = blockDim.x / 2;
	while (i != 0)
	{
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0)
		c[blockIdx.x] = cache[0];
}

extern "C" void Cuda_RayTracing_Init()
{

}
