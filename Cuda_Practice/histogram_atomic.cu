#include "hip/hip_runtime.h"
#include "stdafx.h"

#define SIZE (100 * 1024 * 1024)

__global__ void histo_kernel(unsigned char *buffer, int size, unsigned int *histo)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;

	while (i < size)
	{
		atomicAdd(&histo[buffer[i]], 1);
		i += stride;
	}
}

__global__ void histo_kernel_optimization(unsigned char *buffer, int size, unsigned int *histo)
{
	__shared__ unsigned int temp[256];
	temp[threadIdx.x] = 0;
	__syncthreads();

	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;

	while (i < size)
	{
		atomicAdd(&histo[buffer[i]], 1);
		i += stride;
	}
	__syncthreads();
	atomicAdd(&histo[threadIdx.x], temp[threadIdx.x]);
}

extern "C" void Init_Histogram()
{
	unsigned char *buffer = new unsigned char[SIZE];

	for (int i = 0; i < SIZE; ++i)
	{
		buffer[i] = rand() % 256;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	unsigned char *dev_buffer;
	unsigned int *dev_histo;

	hipMalloc((void**)&dev_buffer, SIZE);
	hipMemcpy(dev_buffer, buffer, SIZE, hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_histo, 256 * sizeof(long));
	hipMemset(dev_histo, 0, 256 * sizeof(int));

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int blocks = prop.multiProcessorCount;
	//cout << "Block ��  : " << blocks << endl;	

	//histo_kernel << <blocks * 2, 256 >> > (dev_buffer, SIZE, dev_histo);
	histo_kernel_optimization << <blocks * 2, 256 >> > (dev_buffer, SIZE, dev_histo);

	unsigned int histo[256];
	hipMemcpy(histo, dev_histo, 256 * sizeof(int), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	//cout << "Time to generate " << elapsedTime << endl;
	printf("Time to generate : %3.1f ms\n", elapsedTime);
	long histoCount = 0;
	for (int i = 0; i < 256; ++i)
	{
		histoCount += histo[i];
	}

	cout << "Histogram Sum : " << histoCount << endl;

	for (int i = 0; i < SIZE; ++i)
	{
		histo[buffer[i]]--;
	}
	for (int i = 0; i < 256; ++i)
	{
		if (histo[i] != 0) cout << "Failure at " << i << endl;
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_histo);
	hipFree(dev_buffer);
	delete[] buffer;
}