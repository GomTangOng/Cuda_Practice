#include "hip/hip_runtime.h"
#include "stdafx.h"

#define SIZE (100 * 1024 * 1024)

__global__ void histo_kernel(unsigned char *buffer, int size, unsigned int *histo)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;

	while (i < size)
	{
		atomicAdd(&histo[buffer[i]], 1);
		i += stride;
	}
}

extern "C" void Init_Histogram()
{
	unsigned char *buffer = new unsigned char[SIZE];

	for (int i = 0; i < SIZE; ++i)
	{
		buffer[i] = rand() % 'z' + 'a';
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	unsigned char *dev_buffer;
	unsigned int *dev_histo;

	hipMalloc((void**)&dev_buffer, SIZE);
	hipMemcpy(dev_buffer, buffer, SIZE, hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_histo, 256 * sizeof(long));
	hipMemset(dev_histo, 0, 256 * sizeof(int));

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int blocks = prop.multiProcessorCount;

	histo_kernel << <blocks * 2, 256 >> > (dev_buffer, SIZE, dev_histo);

	unsigned int histo[256];
	hipMemcpy(histo, dev_histo, 256 * sizeof(int), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	cout << "Time to generate " << elapsedTime << endl;

	long histoCount = 0;
	for (int i = 0; i < 256; ++i)
	{
		histoCount += histo[i];
	}

	cout << "Histogram Sum : " << histoCount << endl;

	for (int i = 0; i < SIZE; ++i)
	{
		histo[buffer[i]]--;
	}
	for (int i = 0; i < 256; ++i)
	{
		if (histo[i] != 0) cout << "Failure at " << i << endl;
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_histo);
	hipFree(dev_buffer);
	delete[] buffer;
}