#include "stdafx.h"

extern "C" void cuda_malloc_test(int size, bool up)
{
	hipEvent_t start, stop;
	int *a, *dev_a;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	a = (int *)malloc(size * sizeof(*a));
	hipMalloc((void**)&dev_a, size * sizeof(*dev_a));

	hipEventRecord(start, 0);
	for (int i = 0; i < 100; ++i)
	{
		if (up)
			hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice);
		else
			hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	free(a);
	hipFree(dev_a);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	if (up)
		cout << "malloc_test(HOST=>DEVICE) : " << elapsedTime << endl;
	else
		cout << "malloc_test(DEVICE=>HOST) : " << elapsedTime << endl;
}

extern "C" void cuda_host_alloc_test(int size, bool up)
{
	hipEvent_t start, stop;
	int *a, *dev_a;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	//a = (int *)malloc(size * sizeof(*a));
	hipHostAlloc((void **)&a, size * sizeof(*a), hipHostMallocDefault);
	hipMalloc((void**)&dev_a, size * sizeof(*dev_a));

	hipEventRecord(start, 0);
	for (int i = 0; i < 100; ++i)
	{
		if (up)
			hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice);
		else
			hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	//free(a);
	hipHostFree(a);
	hipFree(dev_a);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	if (up)
		cout << "host_alloc_test(HOST=>DEVICE) : " << elapsedTime << endl;
	else
		cout << "host_alloc_test(DEVICE=>HOST) : " << elapsedTime << endl;
}


